#include "hip/hip_runtime.h"

#include "Kernels.h"

__device__ __forceinline__ TFloat pownd(TFloat x, int n) {
  if(n == 0) return 1;
  TFloat r = x;
  for(int i = 2; i <= n; i++) r *= x;
  return r;
}

__device__ __forceinline__ TFloat BlattWeisskopfFact(int l, TFloat kTsq, TFloat r) {
  TFloat dx = 1.;
  TFloat r2 = pownd(r/0.1973, 2);
  TFloat z  = kTsq*r2;
  if(l == 1)
    dx = 1. + z;
  if(l == 2)
    dx = 9. + 3.*z + z*z;
  if(l == 3)
    dx = 225. + 45.*z + 6.*z*z + pownd(z, 3);
  if(l == 4)
    dx = 11025. + 1575.*z + 135.*z*z + 10.*pownd(z, 3) + pownd(z, 4);
  if(l == 5)
    dx = 893025. + 99225.*z + 6300.*z*z + 315.*pownd(z, 3) + 15.*pownd(z, 4) + pownd(z, 5);
  if(l == 6)
    dx = 108056025. + 9823275.*z + 496125.*z*z + 18900.*pownd(z, 3) + 630.*pownd(z, 4)
        + 21.*pownd(z,5) + pownd(z,6);
  if(l >= 1) dx = dx/pownd(r2, l);
//      if(l >= 7) dx=dx*((1.+4.*q2)*r2)**(L-6)
  return dx;
}

__device__ __forceinline__ TFloat Rho(TFloat s, int J, TFloat qsq, TFloat r) {
  return 2 * sqrt(qsq/s) * pownd(qsq, J) / BlattWeisskopfFact(J, qsq, r);
}

// Simple Breit Wigner function with constant width
__global__ void Kernels::BWConstW(unsigned nev, ccfloat * bw_rk,
                                  TFloat * s_kchan_k, TFloat M2, TFloat MG) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  bw_rk[i] = 1. / ccfloat(s_kchan_k[i]-M2, MG);
}

// depends on res.M, res.G, res.r
__global__ void Kernels::BWVarW(unsigned nev, ccfloat * bw_rk, TFloat * s_kchan_k,
                                TFloat * v1kT2_k, TFloat M2, TFloat MGRho, TFloat r, unsigned J) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  bw_rk[i] = 1. / ccfloat(s_kchan_k[i]-M2, MGRho*Rho(s_kchan_k[i], J, -v1kT2_k[i], r));
}

__global__ void Kernels::BlWDec(unsigned nev, TFloat * blw_dec_rk,
                                TFloat * v1kT2_k, TFloat r, unsigned J) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  blw_dec_rk[i] = sqrt( BlattWeisskopfFact(J, -v1kT2_k[i], r) );
}

__global__ void Kernels::VarFactor(unsigned nev, ccfloat * var_factor,
                                   ccfloat * bw_rk, TFloat * blw_dec_rk, ccfloat prod_factor) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  var_factor[i] = prod_factor * bw_rk[i] / blw_dec_rk[i];
}

__global__ void Kernels::Ampl(unsigned nev, ccfloat * ampl_rm,
                              ccfloat * precalc_factor_rkm, ccfloat * var_factor) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  ampl_rm[i] += precalc_factor_rkm[i] * var_factor[i];
}

__global__ void Kernels::AddAmpl(unsigned nev, ccfloat * ampl_sum, ccfloat * ampl_rm) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  ampl_sum[i] += ampl_rm[i];
}

__global__ void Kernels::ConvergeAmpl(unsigned nev, TFloat jpsiDens,
                                      TFloat * t_ampl_sq, ccfloat * ampl_sum) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  TFloat re = real(ampl_sum[i]),
         im = imag(ampl_sum[i]);
  t_ampl_sq[i] += jpsiDens * (re*re + im*im);
}

__global__ void Kernels::ConvergeNorm(unsigned nev, TFloat jpsiDens, TFloat * N,
                                      ccfloat * ampl_sum, TFloat * weight) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  TFloat re = real(ampl_sum[i]),
         im = imag(ampl_sum[i]);
  N[i] = jpsiDens * (re*re + im*im) * weight[i];
}

__global__ void Kernels::Log(unsigned nev, TFloat * val, TFloat * ampl_sq, TFloat * weight) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  val[i] = - weight[i] * log(ampl_sq[i]);
}

__global__ void Kernels::ChangeSign(unsigned nev, TFloat * ampl_sq_ipar) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  ampl_sq_ipar[i] *= -1;
}

__global__ void Kernels::F1(unsigned nev, TFloat * F1, TFloat * ampl_sq_ipar,
                            TFloat * ampl_sq, TFloat * weight) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  F1[i] = - weight[i] * ampl_sq_ipar[i] / ampl_sq[i];
}

__global__ void Kernels::F2(unsigned nev, TFloat * F2, TFloat * ampl_sq_ipar,
                            TFloat * ampl_sq_jpar, TFloat * ampl_sq, TFloat * weight) {
  unsigned i = threadIdx.x + blockDim.x*blockIdx.x;
  if(i >= nev) return;
  F2[i] = weight[i] * ampl_sq_ipar[i] * ampl_sq_jpar[i] / (ampl_sq[i]*ampl_sq[i]);
}

__global__ void Kernels::Reduce(unsigned n, TFloat * p_in, TFloat * p_out) {
  extern __shared__ TFloat sdata[];
  // each thread loads one element from global to shared mem
  unsigned int ithr = threadIdx.x;
  unsigned int nthr = blockDim.x;
  unsigned int offset = nthr*2*blockIdx.x;
  unsigned int i = offset + ithr;
  unsigned int j = i + nthr;
  if(j < n)
    sdata[ithr] = p_in[i] + p_in[j];
  else if(i < n)
    sdata[ithr] = p_in[i];
  else
    sdata[ithr] = 0;
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s = nthr/2; s > 1; s >>= 1) {
    if(ithr < s) sdata[ithr] += sdata[ithr+s];
    __syncthreads();
  }
  // write result for this block to global mem
  if (ithr == 0) p_out[blockIdx.x] = sdata[0]+sdata[1];
}
