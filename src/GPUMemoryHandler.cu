
#include "GPUMemoryHandler.h"
#include "Kernels.h"
#include <cstdio>

void Err(hipError_t err, const char * f);

GPUMemoryHandler::GPUMemoryHandler() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  nThr = deviceProp.maxThreadsPerBlock;
}

GPUMemoryHandler * GPUMemoryHandler::Get() {
  static GPUMemoryHandler gInstance;
  return &gInstance;
}

void * GPUMemoryHandler::Allocate(unsigned n_bytes) {
  void * p;
  hipError_t err = hipMalloc(&p, n_bytes);
  Err(err, "Alloc");
  return p;
}

void GPUMemoryHandler::Free(void * p) {
  hipError_t err = hipFree(p);
  Err(err, "Free");
}

void GPUMemoryHandler::Zero(void * p, unsigned n_bytes) {
  hipError_t err = hipMemset(p, 0, n_bytes);
  Err(err, "Zero");
}

void GPUMemoryHandler::Copy(void * to, const void * from, unsigned n_bytes) {
  hipError_t err = hipMemcpy(to, from, n_bytes, hipMemcpyDeviceToDevice);
  Err(err, "Copy");
}

void GPUMemoryHandler::ToGPU(void * p, void * p_d, unsigned nbytes) {
  hipError_t err = hipMemcpy(p_d, p, nbytes, hipMemcpyHostToDevice);
  Err(err, "ToGPU");
}

void GPUMemoryHandler::FromGPU(void * p, void * p_d, unsigned nbytes) {
  hipError_t err = hipMemcpy(p, p_d, nbytes, hipMemcpyDeviceToHost);
  Err(err, "FromGPU");
}

TFloat GPUMemoryHandler::Reduce(TFloat * p_in, unsigned n) {
  while(n > 1) {
    unsigned nBlocks = CalcBlockDim(n);
    TFloat * p_out = _Allocate<TFloat>(nBlocks);
    Kernels::Reduce<<<nBlocks, nThr, nThr*sizeof(TFloat)>>>(n, p_in, p_out);
    Free(p_in);
    p_in = p_out;
    n = nBlocks;
  }
  TFloat ret = 0;
  FromGPU(&ret, p_in, sizeof(TFloat));
  Free(p_in);
  return ret;
}

inline void Err(hipError_t err, const char * f) {
  if(err != hipSuccess) {
    fprintf(stderr, "GPUMemoryHandler::%s: error \"%s\"\n", f, hipGetErrorString(err));
    fflush(stderr);
    exit(-1);
  }
}
